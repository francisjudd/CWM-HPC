// In this assignment you will write a basic kernel where every thread 
// will write out to console string "Hello world!".
// You will also initialize GPU using cudaSetDevice() and also launch
// your "Hello world" kernel.

#include <stdio.h>
#include <stdlib.h>
// we have to include few more things
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

//----------------------------------------------------------------------
// TASK 2: Write a "Hello world" kernel
// 
// Remember that kernel is defined by __global__ and inside it looks like
// a serial code for CPU. For printing out to console you can use printf().

// write your kernel here

__global__ void helloworld_GPU( void )
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  printf("Hello world from thread %d!\n",index);
}

//----------------------------------------------------------------------

int main(void) {
  //----------------------------------------------------------------------
  // TASK 1: Initiate GPU using cudaSetDevice()
  //
  // You can also try to write a check if there is a device with that id,
  // so the code behaves nicely when it fails
  
  // write you GPU initialization here

  //----------------------------------------------------------------------
  // Initiate GPU 
  int deviceId = 0;
  int devCount;

  // Get number of GPU available
  hipGetDeviceCount( &devCount );

  // Check we have enough space
  if( deviceId < devCount ) 
  {
    hipSetDevice( deviceId );
  } else return(1);

  //----------------------------------------------------------------------
  // TASK 3: execute your "Hello world" kernel on 1 block with 5 threads 
  //         using execution configuration syntax.
  // 
  // You may use whatever syntax version you prefer, a simplified one 
  // dimensional or full three dimensional call using dim3 data type.
  
  // execute your "Hello world" kernel here
  
  int Gx=1, Gy=1, Gz=1;
  int Bx=5, By=1, Bz=1;

  dim3 Gd(Gx, Gy, Gz);
  dim3 Bd(Bx, By, Bz);

  helloworld_GPU<<< Gd, Bd >>>();

  //----------------------------------------------------------------------
 
  hipDeviceReset(); 
  return (0);
}
