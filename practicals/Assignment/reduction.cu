//
// Include the usual libraries
//
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>


//
// Include cuda libraries
//
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <hip/hip_runtime_api.h>

// Define constants for the program (number of elements)
#define NUM_ELS 1024

__global__ void reduction(float *d_input, float *d_output)
{
  // Allocate shared memory
  __shared__ float smem_array[NUM_ELS];


  // Get thread and global thread ID
  int tid  = threadIdx.x;
  int bid  = blockIdx.x;
  int gtid = threadIdx.x + blockIdx.x * blockDim.x;

  // Loads data into shared memory
  smem_array[tid] = d_input[gtid];
  __syncthreads();


  // Binary Tree Reduction Per Block
  for (int d = (blockDim.x)/2; d > 0; d /= 2) {
    __syncthreads();  // ensure previous step completed 
    if (tid<d)  {
      smem_array[tid] += smem_array[tid + d];
    }
  }

  // First thread puts result into global memory
  if (tid==0) {
     d_output[bid] = smem_array[0];
  }
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

int main( int argc, const char** argv) 
{
    int num_els, i;
    int num_threads, num_blocks, mem_size;

    // Define  Host Memory Pointers
    float *h_data;
    float *d_input, *d_output;


    // Initialise card
    printf("Enter number of randoms required: ");
    scanf("%d", &num_els);


    // Calculate remainder of num_els and assign extra blocks as required
    div_t result  = div( num_els, NUM_ELS );

    if( result.rem != 0 ) {
      num_blocks  = result.quot + 1;
    } else {
      num_blocks  = result.quot;
    }

    num_threads   = NUM_ELS;
    mem_size      = sizeof(float) * num_els;


    // Allocate host memory to store the input data
    h_data  = (float*) malloc(mem_size);

    // Initialize to integer values between 0 and 1000
    /*for(int i = 0; i < num_els; i++) {
        h_data[i] = 1.0f;
    }*/


    // Allocate device memory input and output arrays
    hipMalloc((void**)&d_input, mem_size);
    hipMalloc((void**)&d_output, num_blocks*sizeof(float));

    
    // Declare variables
    hiprandGenerator_t gen;

    // Create random number generator
    hiprandCreateGenerator( &gen, HIPRAND_RNG_PSEUDO_DEFAULT );

    // Set the generator options
    hiprandSetPseudoRandomGeneratorSeed( gen, 1234ULL );

    // Generate the randoms!!
    hiprandGenerateUniform( gen, d_input, num_els );
    
    // Copy host memory to device input array
    hipMemcpy(h_data, d_input, mem_size, hipMemcpyDeviceToHost);

    for(i = 0; i < num_els;i++) {
      printf("h_data[%d] = %f\n",i,h_data[i]);
    }

    // Execute the kernel
    reduction<<< num_blocks, num_threads>>>(d_input,d_output);


    // Copy result from device to host
    hipMemcpy(h_data, d_output, num_blocks*sizeof(float), hipMemcpyDeviceToHost);

    
    // Sum reductions from each block
    for( i = 1; i < num_blocks; i++ ) {
      h_data[0] += h_data[i];
    }

    // Check results
    printf("Reduction error = %f\n",h_data[0]/num_els);


    // Cleanup memory
    hiprandDestroyGenerator(gen);
    free(h_data);
    hipFree(d_input);
    hipFree(d_output);


    // CUDA exit -- needed to flush printf write buffer
    hipDeviceReset();
}

