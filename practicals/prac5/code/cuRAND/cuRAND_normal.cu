#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hiprand.h>

int main( void ) {

  // Allocate pointers for host and device memory
  float *h_input, *h_output;
  float *d_input, *d_output;


  // Declare variables
  int length of array;



  size_t mem_size;

  // malloc() host memory - our RAM
  h_input  = (float *)malloc( mem_size );
  h_output = (float *)malloc( mem_size );

  // allocate device memory input and output arrays
  hipMalloc((void**)&d_input,  mem_size);
  hipMalloc((void**)&d_output, mem_size);
 
  //
  // Do something here!
  //

    // Copy host memory to device input array
    hipMemcpy(d_input,  h_input,  mem_size, hipMemcpyHostToDevice);

    //
    // Do something on GPU!
    //

      // Declare variables
      hiprandGenerator_t gen;
      
      // Create random number generator
      hiprandCreateGenerator( &gen, HIPRAND_RNG_PSEUDO_DEFAULT) );

      // Set the generator options
      hiprandSetPseudoRandomGeneratorSeed( gen, 1234ULL) );

      // Generate the randoms!!
      hiprandGenerateNormal( gen, d_input, NUM_ELS, 0.0f, 1.0f) );

      // Send randoms to output array
      d_output = d_input;

    // Copy result from device to host
    hipMemcpy(h_output, d_output, mem_size, hipMemcpyDeviceToHost);

  prinf("[%lf

  // cleanup memory
  free(h_input);
  free(h_output);
  hipFree(d_input);
  hipFree(d_output);

}
